#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: ./memcpy_latency <size>" << std::endl;
        return 1;
    }

    int size = std::stoi(argv[1]); 
    // 分配 CPU 内存
    std::cout<<size<<std::endl;
    char* hostSrc = new char[size];
    char* hostDst = new char[size];

    // 分配 GPU 内存
    char* deviceSrc;
    char* deviceDst;
    hipMalloc((void**)&deviceSrc, size);
    hipMalloc((void**)&deviceDst, size);

    // 创建 CUDA 事件对象
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    // 将输入数据从 CPU 复制到 GPU
    hipMemcpy(deviceSrc, hostSrc, size, hipMemcpyHostToDevice);

    // 启动计时
 

    // 执行 cudaMemcpy 操作
  

    // 停止计时
    hipEventRecord(end);
    hipEventSynchronize(end);
    hipMemcpy(hostSrc, deviceSrc, size, hipMemcpyDeviceToDevice);
    // 计算延迟
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);

    // 输出延迟结果
    std::cout << "Memcpy Latency: " << milliseconds*1000 << " mu" << std::endl;

    // 释放内存和事件对象
    delete[] hostSrc;
    delete[] hostDst;
    hipFree(deviceSrc);
    hipFree(deviceDst);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return 0;
}
