
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <fstream>

void writelog(int blocknum,int threadnum,int memtransaction,double bandwidth)
{
    std::ofstream outputFile("testlog.txt",std::ios::app);
	if(outputFile.is_open());
	{
		outputFile<<blocknum<<" "<<threadnum<<" "<<memtransaction<<" "<<bandwidth<<std::endl;
	}
}
int main() {
    int N = pow(2,18);

    float* src;
    float* dst;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // 分配 CPU 内存
    float* hostSrc = new float[N];
    float* hostDst = new float[N];
    float elapsedTime = 0;

    // 初始化输入数据
    for (int i = 0; i < N; i++) {
        hostSrc[i] = i;
    }

    // 分配 GPU 内存
    hipMalloc((void**)&src, N * sizeof(float));
    hipMalloc((void**)&dst, N * sizeof(float));
 
    hipMemcpy(src, hostSrc, N * sizeof(float), hipMemcpyHostToDevice);
  
    hipEventRecord(start);
   
    hipMemcpy(hostDst, dst, N * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // 计算运行时间

    // 计算带宽
    double bandwidth= (2*N * sizeof(float)) / (elapsedTime * 1e6);
    std::cout << "Memory Bandwidth: " << bandwidth<< " GB/s" << std::endl;

    
    // 释放 CPU 内存
    delete[] hostSrc;
    delete[] hostDst;

    // 释放 GPU 内存
    hipFree(src);
    hipFree(dst);
    return 0;
}
