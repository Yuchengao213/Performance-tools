
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <fstream>
int main(int argc, char* argv[]) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " N" << std::endl;
        return 1;
    }

    int N = std::atoi(argv[1]);

    float* src;
    float* dst;

    hipEvent_t starth2d, startd2h,stoph2d,stopd2h;
    hipEventCreate(&starth2d);
    hipEventCreate(&stoph2d);
    hipEventCreate(&startd2h);
    hipEventCreate(&stopd2h);
    float* hostSrc = new float[N];
    float* hostDst = new float[N];
    float elapsedTimeh2d = 0;
    float elapsedTimed2h=0;
    for (int i = 0; i < N; i++) {
        hostSrc[i] = i;
    }

    hipMalloc((void**)&src, N * sizeof(float));
    hipMalloc((void**)&dst, N * sizeof(float));
 
    hipEventRecord(starth2d);
    hipMemcpy(src, hostSrc, N * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stoph2d);
    hipEventSynchronize(stoph2d);
    hipEventElapsedTime(&elapsedTimeh2d, starth2d, stoph2d);
    
    hipEventRecord(startd2h);
   
    hipMemcpy(hostDst, dst, N * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stopd2h);
    hipEventSynchronize(stopd2h);
    hipEventElapsedTime(&elapsedTimed2h, startd2h, stopd2h);
    
    double bandwidth= (N * sizeof(float)) / (elapsedTimeh2d * 1e6);
    std::cout << "Host to device copy Bandwidth: " << bandwidth<< " GB/s" << std::endl;
    double bandwidth_d2h= (N * sizeof(float)) / (elapsedTimed2h * 1e6);
    std::cout << "Device to host Bandwidth: " << bandwidth_d2h<< " GB/s" << std::endl;

    delete[] hostSrc;
    delete[] hostDst;

    hipFree(src);
    hipFree(dst);
    return 0;
}
